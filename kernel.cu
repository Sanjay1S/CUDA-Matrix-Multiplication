
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 0

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE]; 
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float Pvalue = 0.0f;

   int i = 0;
   while (i < (k - 1) / TILE_SIZE + 1) {
    if (row < m && i * TILE_SIZE + threadIdx.x < k)
        A_shared[threadIdx.y][threadIdx.x] = A[row * k + i * TILE_SIZE + threadIdx.x];
    else
        A_shared[threadIdx.y][threadIdx.x] = 0.0;

    if (i * TILE_SIZE + threadIdx.y < k && col < n)
        B_shared[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * n + col];
    else
        B_shared[threadIdx.y][threadIdx.x] = 0.0;

    __syncthreads();

    if (row < m && col < n) {
        int j = 0;
        while (j < TILE_SIZE) {
            Pvalue += A_shared[threadIdx.y][j] * B_shared[j][threadIdx.x];
            j++;
        }
    }

    __syncthreads();
    i++;

    }

    if (row < m && col < n) {
        C[row * n + col] = Pvalue;
    }
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((n-1) / BLOCK_SIZE + 1, ((m-1)/BLOCK_SIZE+1));
    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------
    mysgemm<<<dimGrid, dimBlock>>>(m,n,k,A,B,C);
    /*************************************************************************/
    //INSERT CODE HERE

	hipDeviceSynchronize();
    /*************************************************************************/
}


